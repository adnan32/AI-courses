
//Task 2 - Odd-Even Sort using CUDA Adnan Altukleh & Abdulkarim Dawalibi

#include <vector>
#include <algorithm>
#include <iostream>
#include <chrono>
#include <hip/hip_runtime.h>
#define THREADS 1024



__global__ void OES_kernel(int* data, int size, int phase) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;// Get the index of the current element to process
    int thread_offset = index * 2 + phase;
    for (int j = thread_offset;  j < size-1; j += THREADS *2) {
        if (data[j] > data[j + 1]) {
            int temp = data[j];
            data[j] = data[j + 1];
            data[j + 1] = temp;
        }
    }
    
}





void oddevensort(int* x, int size) {
    // Allocate GPU memory
    int *x_d;
    hipMalloc((void**) &x_d, size*sizeof(int));
    // Copy data to GPU memory
    hipMemcpy(x_d, x, size*sizeof(int), hipMemcpyHostToDevice);
    // Perform computation on GPU
    int numThreadsPerBlock = THREADS; // A common choice for the number of threads per block
    int numBlocks = 10;
    for (int i=1; i<=size; i++) {
        OES_kernel<<<numBlocks,numThreadsPerBlock>>>(x_d,size,i%2);
        // Wait for the GPU to finish
    }
    hipDeviceSynchronize();
    // Copy data from GPU memory
    hipMemcpy(x, x_d, size*sizeof(int), hipMemcpyDeviceToHost);
    // Deallocate GPU memory
    hipFree(x_d);
}

void print_sort_status(std::vector<int> numbers)
{
    std::cout << "The input is sorted?: " << (std::is_sorted(numbers.begin(), numbers.end()) == 0 ? "False" : "True") << std::endl;
}

int main(){
    constexpr unsigned int size = 100000; // Number of elements in the input
    // Initialize a vector called numbers with integers of value 0 and size of size above
    std::vector<int> numbers(size);

    // Populate our vector with (pseudo)random numbers
    srand(static_cast<unsigned int>(time(0))); // Seed the random number generator
    generate(numbers.begin(), numbers.end(), rand);
    auto start = std::chrono::steady_clock::now();  // Start the timer
    // Sort using CUDA
    oddevensort(numbers.data(), numbers.size());
    // check if the vector is sorted
    auto end = std::chrono::steady_clock::now();  // End the timer
    print_sort_status(numbers);
    std::cout << "Elapsed time =  " << std::chrono::duration<double>(end - start).count() << " sec\n"; // Print the elapsed time
    return 0;
}